#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>

#include <iostream>
#include <vector>
#include <complex>

#include "../include/hpc_helpers.hpp"

__global__ void do_stuff(thrust::complex<float>* c, int N)
{
    int idx = blockDim.x*blockIdx.x + threadIdx.x;


    if(idx<N) {
        //printf("hello\n");
        c[idx] += thrust::complex<float>(5.0,-1.0);
    }
}

int main(void)
{

    std::vector<std::complex<float> > H(1000);

    int N = 50;
    for(int i=0; i<H.size(); ++i) {
        H[i].real((float)i/N);
        H[i].imag(-i%N);
    }
    
    for(int i=0; i<H.size(); ++i)
		std::cout << H[i] << std::endl;

    thrust::host_vector<thrust::complex<float> > h2(H.begin(), H.end());
    

    //~ for(int i=0; i<h2.size(); ++i)
        //~ h2[i]=H[i];

    //~ std::vector<int > H(1000);

    //~ int N = 50;
    //~ for(int i=0; i<H.size(); ++i) {
        //~ H[i] = i%N;
    //~ }

    //~ thrust::complex<float>* dev;

    //~ std::cout << "malloc" << std::endl;
	//~ hipMalloc(&dev, h2.size()*sizeof(thrust::complex<float>));   		CUERR

    //~ std::cout << "copy" << std::endl;
    //~ thrust::copy(h2.begin(), h2.end(), thrust::device_ptr<thrust::complex<float> >(dev));                      CUERR

   //~ // std::cout << "cast" << std::endl;
   //~ // std::complex<float>* std_dev = (std::complex<float>*) dev;




    //~ std::cout << "declare" << std::endl;
    //~ thrust::device_vector<thrust::complex<float> > D(h2.begin(), h2.end()); CUERR //D(H.begin(), H.end()); CUERR

    //~ std::cout << "cast" << std::endl;
    //~ thrust::complex<float>* dev = thrust::raw_pointer_cast(D.data());

    //std::complex<float>* std_dev = (std::complex<float>*) dev;
    
    std::complex<float>* res;
    hipHostMalloc(&res, H.size()*sizeof(thrust::complex<float>));		CUERR
    
    for(int i=0; i<H.size(); ++i)
		res[i] = thrust::complex<float>(-1.0,-1.0);
    
    std::complex<float>* dev;
    hipMalloc(&dev, H.size()*sizeof(thrust::complex<float>));	CUERR
	hipMemcpy(dev, H.data(), sizeof(thrust::complex<float>)*H.size(), H2D);					CUERR	
	
	thrust::device_ptr<thrust::complex<float>> dev_thrust = thrust::device_pointer_cast((thrust::complex<float>*) dev);  
	thrust::fill(dev_thrust, dev_thrust+H.size(), thrust::complex<float>(-100,-100));			CUERR

    std::cout << H.size() << std::endl;

    int threads = 256;
    int blocks = SDIV(H.size(), threads);
    //do_stuff<<<threads, blocks >>>((thrust::complex<float>*)std_dev, H.size());           CUERR
    do_stuff<<<threads, blocks >>>((thrust::complex<float>*)dev, H.size());           CUERR
    std::cout << "end" << std::endl;
    
    hipMemcpy(res, dev, H.size()*sizeof(thrust::complex<float>), D2H);					CUERR
    std::cerr << "dump?" << std::endl;
    
    for(int i=0; i<H.size(); ++i)
		std::cout << res[i] << std::endl;
		//std::cout << h2[i] << std::endl;
    
    //hipFree(std_dev);		CUERR

    return 0;
}
