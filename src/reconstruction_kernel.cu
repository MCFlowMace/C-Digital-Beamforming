#include "hip/hip_runtime.h"
/*
 * reconstruction_kernel.cu
 *
 * Copyright 2020 Florian Thomas <>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 *
 *
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <thrust/device_ptr.h>

#include "hpc_helpers.hpp"
#include "data_packet.hpp"
#include "reconstruction.hpp"


#ifdef USE_GPU

//not very nice solution
#include "../src/reconstruction.cpp"

template <typename value_t>
__global__ void reconstruction(thrust::complex<value_t>* samples,
                                thrust::complex<value_t>* grid_phase,
                                value_t* coords, value_t* rec, value_t R,
                                int bins, int grid_size, int N)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {

        int i = tid%grid_size;
        int j = (tid/grid_size)%grid_size;
        int k = tid/(grid_size*grid_size);

        value_t x = coords[j];
        value_t y = coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {

            thrust::complex<value_t> accum(0);

            for(int l=0; l<N; ++l) {
                //accum += samples[l*bins+k]*grid_phase[((grid_size*i+j)*bins+k)*N+l]; //not coalesced
                accum += samples[l*bins+k]*grid_phase[((l*bins+k)*grid_size+j)*grid_size+i];
            }
            rec[(grid_size*i+j)*bins+k] = thrust::abs(accum);

        }

    }
}

template <typename value_t>
void Reconstruction<value_t>::calc_phase(
                        const std::vector<arma::Mat<value_t>>& grid_time_delays,
                        const std::vector<arma::Mat<value_t>>& grid_phis,
                        std::complex<value_t>* const grid_phase)
{
	
	int bins = frequency.n_elem;

    for(int i=0; i<N; ++i) {
        for(int l=0; l<bins; ++l) {
            for(int k=0; k<grid_size; ++k) {
                for(int j=0; j<grid_size; ++j) {
                    value_t phi = grid_time_delays[i](k,j)*(2*M_PI*frequency(l)+wmix);
                    phi += grid_phis[i](k,j);
                    //grid_phase[((grid_size*j+k)*frequency.n_elem+l)*N+i] = std::complex<value_t>(cos(phi), sin(phi));
					grid_phase[((i*bins+l)*grid_size+k)*grid_size+j] = std::complex<value_t>(cos(phi), sin(phi));
                }
            }
        }
    }

}

template <typename value_t>
void Reconstruction<value_t>::set_grid_phase(std::complex<value_t>** grid_phase)
{
    hipMalloc(&(this->grid_phase),
        N*grid_size*grid_size*frequency.n_elem*sizeof(std::complex<value_t>)); CUERR

    hipMemcpy(this->grid_phase, *grid_phase,
        N*grid_size*grid_size*frequency.n_elem*sizeof(std::complex<value_t>),
        H2D);                           CUERR
}

template <typename value_t>
void Reconstruction<value_t>::free_grid_phase()
{
    if(grid_phase) {
        hipFree(grid_phase);   CUERR
        grid_phase=nullptr;
    }
}

template <typename value_t>
void Reconstruction<value_t>::run(const std::vector<Data_Packet<value_t>>& samples)
{


    TIMERSTART(REC)

    int bins = samples[0].frequency.n_elem;

    //reorder data
    thrust::host_vector<thrust::complex<value_t> > samples_H(N*bins, thrust::complex<value_t>(0,1));   CUERR

    TIMERSTART(COPY_CPU)
    for(int i=0; i<samples.size(); ++i) {
        thrust::copy(samples[i].frequency_data.begin(),
                        samples[i].frequency_data.end(),
                        samples_H.begin()+i*bins);                      CUERR
    }
    TIMERSTOP(COPY_CPU)

    //copy data to GPU
    TIMERSTART(COPY_GPU)
    thrust::device_vector<thrust::complex<value_t> > samples_D = samples_H;  CUERR
                                                            //(
                                                            //samples_H.begin(),
                                                            //samples_H.end()); CUERR

    thrust::device_vector<value_t> coords_D(grid.coords.begin(),
                                            grid.coords.end());         CUERR
    TIMERSTOP(COPY_GPU)

    thrust::device_vector<value_t> reconstructed_D(grid_size*grid_size*bins);
    thrust::fill(reconstructed_D.begin(), reconstructed_D.end(), value_t(-1));

    thrust::complex<value_t>* samples_dev = thrust::raw_pointer_cast(samples_D.data());
    value_t* rec_dev = thrust::raw_pointer_cast(reconstructed_D.data());
    value_t* coords_dev = thrust::raw_pointer_cast(coords_D.data());

    int threads = 512;
    int tasks=grid_size*grid_size*bins;
    int blocks = SDIV(tasks, threads);

    std::cerr << bins << " " << grid_size << " " << N << std::endl;

    TIMERSTART(KERNEL)
    reconstruction<<<blocks, threads>>>(samples_dev,
                                        (thrust::complex<value_t>*)grid_phase,
                                        coords_dev, rec_dev, grid.R,
                                        bins, grid_size, N);   CUERR
    TIMERSTOP(KERNEL)

    //~ thrust::host_vector<value_t> reconstructed_H(reconstructed_D);

    //~ int count=0;
    //~ for(auto i = reconstructed_H.begin(); i!=reconstructed_H.end(); i++) {

        //~ if(*i==0)
            //~ count++;
    //~ }
    //~ std::cout << "zeros: " << count << std::endl;

    //copy back result
    TIMERSTART(COPY_BACK)
    thrust::copy(reconstructed_D.begin(), reconstructed_D.end(),
                    reconstructed.begin());                             CUERR
	TIMERSTOP(COPY_BACK)

    TIMERSTOP(REC)
}

DEFINE_TEMPLATES(Reconstruction)

#endif
