#include "hip/hip_runtime.h"
/*
 * reconstruction_kernel.cu
 *
 * Copyright 2020 Florian Thomas <>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 *
 *
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <thrust/device_ptr.h>

#include "hpc_helpers.hpp"
#include "data_packet.hpp"
#include "reconstruction.hpp"

//speed of light in cm/s
#define SPEED_OF_LIGHT 29979245800 


#ifdef USE_GPU

//not very nice solution
#include "../src/reconstruction.cpp"

template <typename value_t>
__global__ void reconstruction(thrust::complex<value_t>* samples,
                                thrust::complex<value_t>* grid_phase,
                                value_t* coords, value_t* rec, value_t R,
								int bins, int grid_size, int N)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {

        int i = tid%grid_size;
        int j = (tid/grid_size)%grid_size;
        int k = tid/(grid_size*grid_size);

        value_t x = coords[j];
        value_t y = coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {

            thrust::complex<value_t> accum(0);

            for(int l=0; l<N; ++l) {
                accum += samples[l*bins+k]*grid_phase[((l*bins+k)*grid_size+j)*grid_size+i];
            }
            rec[(grid_size*i+j)*bins+k] = thrust::abs(accum);

        }

    }
}

#define NANTENNAS 30
 /*
template <typename value_t>
__global__ void reconstruction_red(thrust::complex<value_t>* samples,
                                value_t* frequencies, value_t* time_delays,
                                value_t* phis, value_t* rec, value_t R,
                                value_t wmix, int bins, int grid_size, int N)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {

        int i = tid%grid_size;
        int j = (tid/grid_size)%grid_size;
        int k = tid/(grid_size*grid_size);

        value_t x = (-1+(value_t)(2*j+1)/grid_size)*R; //coords[j];
        value_t y = (-1+(value_t)(2*i+1)/grid_size)*R; //coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {
			
			value_t fc = 2*M_PI*frequencies[k] + wmix; //maybe to shared memory
			//value_t f = wmix/2;

            thrust::complex<value_t> accum(0);
            int index_0 = grid_size*j+i;
            int grid_size2 = grid_size*grid_size;

			#pragma unroll
            for(int l=0; l<NANTENNAS; ++l) {
				//~ value_t phi = time_delays[(l*grid_size+j)*grid_size+i]*fc;
				//~ phi += phis[(l*grid_size+j)*grid_size+i];
				
				int index_l = grid_size2*l+index_0;
				value_t phi = time_delays[index_l]*fc + phis[index_l];
				
				//value_t phi = ((l*grid_size+j)*grid_size+i)*(2*M_PI*f+wmix);
				//phi += (l*grid_size+j)*grid_size+i;
				//value_t phi2 = phi*phi;
				//value_t phi3 = phi2*phi;
				
				//~ value_t real;
				//~ value_t imag;
				//~ __sincosf(phi, &imag, &real);
				//~ thrust::complex<value_t> phase(real, imag);
				thrust::complex<value_t> phase(__cosf(phi), __sinf(phi));
				
				//thrust::complex<value_t> phase(time_delays[index_l], phis[index_l]);

                accum += samples[l*bins+k]*phase;
            }
            rec[(grid_size*i+j)*bins+k] = thrust::norm(accum);
            //rec[(grid_size*i+j)*bins+k] = accum.real()*accum.real() + accum.imag()*accum.imag();

        }

    }
} */

template <typename value_t>
__global__ void reconstruction_red(thrust::complex<value_t>* samples,
                                value_t* frequencies, value_t* time_delays,
                                value_t* phis, value_t* rec, value_t R,
                                value_t wmix, int bins, int grid_size, int N,
                                int packet)
{
	
	/*
	 * Fastest so far
	 * 
	 * */
	
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {
		
		int k = tid%bins;
        int i = (tid/bins)%grid_size;
        int j = tid/(bins*grid_size);

        value_t x = (-1+(value_t)(2*j+1)/grid_size)*R; //coords[j];
        value_t y = (-1+(value_t)(2*i+1)/grid_size)*R; //coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {
			
			value_t fc = 2*M_PI*frequencies[k] + wmix; //maybe to shared memory
			//value_t f = wmix/2;

            thrust::complex<value_t> accum(0);

            for(int l=0; l<N; ++l) {
				value_t phi = time_delays[(j*grid_size+i)*N+l]*fc + phis[(j*grid_size+i)*N+l];
				
				thrust::complex<value_t> phase(__cosf(phi), __sinf(phi));
				
				//thrust::complex<value_t> phase(time_delays[(j*grid_size+i)*N+l]*fc, phis[(j*grid_size+i)*N+l]);

                accum += samples[(packet*N+l)*bins+k]*phase;
            }
            rec[((packet*grid_size+i)*grid_size+j)*bins+k] = thrust::norm(accum);

        }

    }
}

template <typename value_t>
__global__ void reconstruction_red2(thrust::complex<value_t>* samples,
                                value_t* frequencies,
                                value_t* coords, value_t* rec, value_t R,
                                value_t wmix, int bins, int grid_size, int N)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {

        int i = tid%grid_size;
        int j = (tid/grid_size)%grid_size;
        int k = tid/(grid_size*grid_size);

        value_t x = coords[j];
        value_t y = coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {
			
			value_t f = frequencies[k]; //maybe to shared memory

            thrust::complex<value_t> accum(0);

            for(int l=0; l<N; ++l) {
				value_t angle = frequencies[l]; //(value_t)l/N*2*M_PI;
				value_t xi = R*cos(angle);
				value_t yi = R*sin(angle);
				
				value_t dx = xi-x;
				value_t dy = yi-y;
				value_t dist = sqrt(dx*dx + dy*dy);
				
				value_t time_delay = dist/SPEED_OF_LIGHT;
				value_t phi_c = atan2(-dy, -dx) + M_PI;
				
				//value_t phi = time_delays[(l*grid_size+j)*grid_size+i]*(2*M_PI*f+wmix);
				//phi += phis[(l*grid_size+j)*grid_size+i];
				
				value_t phi = time_delay*(2*M_PI*f+wmix)+phi_c;
				thrust::complex<value_t> phase(cos(phi), sin(phi));
                accum += samples[l*bins+k]*phase;
            }
            rec[(grid_size*i+j)*bins+k] = thrust::abs(accum);

        }

    }
}

template <typename value_t>
void Reconstruction<value_t>::calc_phase(
                        const std::vector<arma::Mat<value_t>>& grid_time_delays,
                        const std::vector<arma::Mat<value_t>>& grid_phis,
                        std::complex<value_t>* const grid_phase)
{
	
	int bins = frequency.n_elem;

    for(int i=0; i<N; ++i) {
		arma::Mat<value_t> time_delay = grid_time_delays[i].t();
		arma::Mat<value_t> phi_c = grid_phis[i].t();
        for(int l=0; l<bins; ++l) {
			value_t f = frequency(l);
            for(int k=0; k<grid_size; ++k) {
                for(int j=0; j<grid_size; ++j) {
                    //~ value_t phi = grid_time_delays[i](j,k)*(2*M_PI*frequency(l)+wmix);
                    //~ phi += grid_phis[i](j,k);
                    value_t phi = time_delay(j,k)*(2*M_PI*f+wmix);
                    phi += phi_c(j,k);
                    //grid_phase[((grid_size*j+k)*frequency.n_elem+l)*N+i] = std::complex<value_t>(cos(phi), sin(phi));
					grid_phase[((i*bins+l)*grid_size+k)*grid_size+j] = std::complex<value_t>(cos(phi), sin(phi));
                }
            }
        }
    }
}

template <typename value_t>
void Reconstruction<value_t>::set_grid_phase(std::complex<value_t>** grid_phase)
{
    //~ hipMalloc(&(this->grid_phase),
        //~ N*grid_size*grid_size*frequency.n_elem*sizeof(std::complex<value_t>)); CUERR

    //~ hipMemcpy(this->grid_phase, *grid_phase,
        //~ N*grid_size*grid_size*frequency.n_elem*sizeof(std::complex<value_t>),
        //~ H2D);                           CUERR
}

template <typename value_t>
void Reconstruction<value_t>::free_grid_phase()
{
    if(grid_phase) {
        hipFree(grid_phase);   CUERR
        grid_phase=nullptr;
    }
}

template <typename value_t>
void Reconstruction<value_t>::init_gpu()
{

    thrust::host_vector<value_t> time_delays_H(N*grid_size*grid_size);	CUERR
    thrust::host_vector<value_t> phis_H(N*grid_size*grid_size);			CUERR

	for(int i=0; i<N; ++i) {
		arma::Mat<value_t> mat_delay= grid_time_delays[i].t();
		arma::Mat<value_t> mat_phi = grid_phis[i].t();
		thrust::copy(mat_delay.begin(),mat_delay.end(),
					time_delays_H.begin()+i*grid_size*grid_size);		CUERR
		thrust::copy(mat_phi.begin(),mat_phi.end(),
					phis_H.begin()+i*grid_size*grid_size);		CUERR
	}
	
								
	hipMalloc(&(this->frequencies_dev), frequency.n_elem*sizeof(value_t)); CUERR

	hipMalloc(&(this->time_delays_dev), N*grid_size*grid_size*sizeof(value_t)); CUERR

	hipMalloc(&(this->phis_dev), N*grid_size*grid_size*sizeof(value_t)); CUERR

	
	//hipMemcpy(dev_test, test.data(), test.size(), H2D);		CUERR
	
	std::cerr << "copy2" << std::endl;
	//~ thrust::copy(time_delays_H.begin(), time_delays_H.end(), 
											//~ this->time_delays_dev);		CUERR
	hipMemcpy(this->time_delays_dev, time_delays_H.data(), 
										time_delays_H.size(), H2D);		CUERR
	
	std::cerr << "copy3" << std::endl;				
	//~ thrust::copy(phis_H.begin(), phis_H.end(), 
											//~ this->phis_dev);		CUERR
											
	hipMemcpy(this->phis_dev, phis_H.data(), 
										phis_H.size(), H2D);		CUERR
	
	std::cerr << "copy1" << std::endl;
	//~ thrust::copy(this->frequency.begin(),this->frequency.end(), 
											//~ this->frequencies_dev);		CUERR
											
	hipMemcpy(this->frequencies_dev, this->frequency.memptr(), 
										this->frequency.size(), H2D);		CUERR
}

#define FREE(pointer) \
    do { \
		if(pointer) { \
			hipFree(pointer); \
			CUERR \
			pointer=nullptr; \
		} \
    } while (false)  

template <typename value_t>
void Reconstruction<value_t>::free_gpu()
{
	FREE(grid_phase);
	FREE(frequencies_dev);
	FREE(time_delays_dev);
	FREE(phis_dev);
}

template <typename value_t>
void Reconstruction<value_t>::run(const std::vector<std::vector<Data_Packet<value_t>>>& samples)
{
	std::cerr << "run" << std::endl;

    TIMERSTART(REC)

    int bins = samples[0][0].frequency.n_elem;
    int n_packets = samples[0].size();
    std::cerr << "packets: " << n_packets << " antennas: " << samples.size() << std::endl;
    std::cerr << samples.size() << " " << samples[0].size() << std::endl;
    
    
    //~ for(int j=0; j<n_packets; ++j) {

        //~ std::vector<Data_Packet<float>> data_in;

        //~ for(int i=0; i<data_out.size(); ++i)
            //~ data_in.push_back(data_out[i][j]);

    //~ }

    //reorder data
    thrust::host_vector<thrust::complex<value_t> > samples_H(n_packets*N*bins, 
										thrust::complex<value_t>(0,1));   CUERR
										
	std::cerr << "Copy CPU" << std::endl;
	size_t memSize=sizeof(thrust::complex<value_t>)*n_packets*N*bins;
    TIMERSTART(COPY_CPU)
    for(int j=0; j<n_packets; ++j) {
		for(int i=0; i<N; ++i) {
			//std::cerr << i << " " << j << " " << samples[i][j].frequency_data[0] << std::endl;
			thrust::copy(samples[i][j].frequency_data.begin(),
							samples[i][j].frequency_data.end(),
							samples_H.begin()+(j*N+i)*bins);                      CUERR
		}
	}
    //time_delays[(l*grid_size+j)*grid_size+i]
    //TIMERSTOP(COPY_CPU)
    TIMERBW(memSize, COPY_CPU)

    //copy data to GPU
    
    std::cerr << "Copy GPU" << std::endl;
    TIMERSTART(COPY_GPU)
    thrust::device_vector<thrust::complex<value_t> > samples_D = samples_H;  CUERR
                                                            //(
                                                            //samples_H.begin(),
                                                            //samples_H.end()); CUERR

    //thrust::device_vector<value_t> coords_D(grid.coords.begin(),
    //                                        grid.coords.end());         CUERR
                                            

    //TIMERSTOP(COPY_GPU)
    TIMERBW(memSize, COPY_GPU)

    thrust::device_vector<value_t> reconstructed_D(grid_size*grid_size*bins*n_packets);
    thrust::fill(reconstructed_D.begin(), reconstructed_D.end(), value_t(-1));

    thrust::complex<value_t>* samples_dev = thrust::raw_pointer_cast(samples_D.data());
    value_t* rec_dev = thrust::raw_pointer_cast(reconstructed_D.data());
    //value_t* coords_dev = thrust::raw_pointer_cast(coords_D.data());
    //~ value_t* time_delays_dev = thrust::raw_pointer_cast(time_delays_D.data());
	//~ value_t* phis_dev = thrust::raw_pointer_cast(phis_D.data());
	//~ value_t* frequencies_dev = thrust::raw_pointer_cast(frequencies_D.data());
	
    int threads = 512;
    int tasks=grid_size*grid_size*bins;
    int blocks = SDIV(tasks, threads);


    //~ TIMERSTART(KERNEL)
    //~ reconstruction<<<blocks, threads>>>(samples_dev,
                                        //~ (thrust::complex<value_t>*)grid_phase,
                                        //~ coords_dev, rec_dev, grid.R,
                                        //~ bins, grid_size, N);   CUERR
    //~ TIMERSTOP(KERNEL)
    
    TIMERSTART(KERNELS)
    for(int packet=0; packet<n_packets; ++packet) {
		std::cerr << packet << std::endl;
		TIMERSTART(KERNEL_RED)
		reconstruction_red<<<blocks, threads>>>(samples_dev,
											frequencies_dev, time_delays_dev,
											phis_dev, rec_dev, grid.R,
											wmix, bins, grid_size, N, packet);   CUERR
		TIMERSTOP(KERNEL_RED)
	}
	TIMERSTOP(KERNELS)

    //copy back result
    size_t memsize_res = grid_size*grid_size*bins*n_packets*sizeof(value_t);
    TIMERSTART(COPY_BACK)
    thrust::copy(reconstructed_D.begin(), reconstructed_D.end(),
                    reconstructed.begin());                             CUERR
	//TIMERSTOP(COPY_BACK)
	TIMERBW(memsize_res, COPY_BACK)

    TIMERSTOP(REC)
}

//DEFINE_TEMPLATES(Reconstruction)
template class Reconstruction<float>;

#endif
