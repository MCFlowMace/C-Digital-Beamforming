#include "hip/hip_runtime.h"
/*
 * reconstruction_kernel.cu
 *
 * Copyright 2020 Florian Thomas <>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 *
 *
 */
#include <stdexcept>
#include <cfloat>

#include <thrust/complex.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "hpc_helpers.hpp"
#include "data_packet.hpp"
#include "reconstruction_gpu.hpp"
#include "utility_macros.hpp"

//speed of light in cm/s
#define SPEED_OF_LIGHT 29979245800


template <typename value_t>
__global__ void reconstruction(thrust::complex<value_t>* samples,
                                thrust::complex<value_t>* grid_phase,
                                value_t* coords, value_t* rec, value_t R,
								int bins, int grid_size, int N)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {

        int i = tid%grid_size;
        int j = (tid/grid_size)%grid_size;
        int k = tid/(grid_size*grid_size);

        value_t x = coords[j];
        value_t y = coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {

            thrust::complex<value_t> accum(0);

            for(int l=0; l<N; ++l) {
                accum += samples[l*bins+k]*grid_phase[((l*bins+k)*grid_size+j)*grid_size+i];
            }
            rec[(grid_size*i+j)*bins+k] = thrust::abs(accum);

        }

    }
}

#define NANTENNAS 30
 /*
template <typename value_t>
__global__ void reconstruction_red(thrust::complex<value_t>* samples,
                                value_t* frequencies, value_t* time_delays,
                                value_t* phis, value_t* rec, value_t R,
                                value_t wmix, int bins, int grid_size, int N)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {

        int i = tid%grid_size;
        int j = (tid/grid_size)%grid_size;
        int k = tid/(grid_size*grid_size);

        value_t x = (-1+(value_t)(2*j+1)/grid_size)*R; //coords[j];
        value_t y = (-1+(value_t)(2*i+1)/grid_size)*R; //coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {
			
			value_t fc = 2*M_PI*frequencies[k] + wmix; //maybe to shared memory
			//value_t f = wmix/2;

            thrust::complex<value_t> accum(0);
            int index_0 = grid_size*j+i;
            int grid_size2 = grid_size*grid_size;

			#pragma unroll
            for(int l=0; l<NANTENNAS; ++l) {
				//~ value_t phi = time_delays[(l*grid_size+j)*grid_size+i]*fc;
				//~ phi += phis[(l*grid_size+j)*grid_size+i];
				
				int index_l = grid_size2*l+index_0;
				value_t phi = time_delays[index_l]*fc + phis[index_l];
				
				//value_t phi = ((l*grid_size+j)*grid_size+i)*(2*M_PI*f+wmix);
				//phi += (l*grid_size+j)*grid_size+i;
				//value_t phi2 = phi*phi;
				//value_t phi3 = phi2*phi;
				
				//~ value_t real;
				//~ value_t imag;
				//~ __sincosf(phi, &imag, &real);
				//~ thrust::complex<value_t> phase(real, imag);
				thrust::complex<value_t> phase(__cosf(phi), __sinf(phi));
				
				//thrust::complex<value_t> phase(time_delays[index_l], phis[index_l]);

                accum += samples[l*bins+k]*phase;
            }
            rec[(grid_size*i+j)*bins+k] = thrust::norm(accum);
            //rec[(grid_size*i+j)*bins+k] = accum.real()*accum.real() + accum.imag()*accum.imag();

        }

    }
} */

template <typename value_t>
__device__ inline value_t weighted_beamforming(value_t const fc, int const N, 
												int const grid_size, 
												int const packet, int const bins,
												int const i, int const j, int const k,
												value_t const * const time_delays,
												value_t const * const phis,
												thrust::complex<value_t> const * const samples)
{
	thrust::complex<value_t> accum(0);
	value_t A{0};
	for(int l=0; l<N; ++l) {
		value_t t = time_delays[(j*grid_size+i)*N+l];
		value_t phi = t*fc + phis[(j*grid_size+i)*N+l];
		thrust::complex<value_t> phase(__cosf(phi), __sinf(phi));
		
		A += 1/(t*t);

		accum += samples[(packet*N+l)*bins+k]*phase/t;
		
	}
	return thrust::norm(accum)*SPEED_OF_LIGHT/A;
}

template <typename value_t>
__device__ inline value_t beamforming(value_t const fc, int const N, 
										int const grid_size, 
										int const packet, int const bins,
										int const i, int const j, int const k,
										value_t const * const time_delays,
										value_t const * const phis,
										thrust::complex<value_t> const * const samples)
{
	
	thrust::complex<value_t> accum(0);
	for(int l=0; l<N; ++l) {
		value_t phi = time_delays[(j*grid_size+i)*N+l]*fc + phis[(j*grid_size+i)*N+l];
		
		thrust::complex<value_t> phase(__cosf(phi), __sinf(phi));
		
		//thrust::complex<value_t> phase(time_delays[(j*grid_size+i)*N+l]*fc, phis[(j*grid_size+i)*N+l]);

		accum += samples[(packet*N+l)*bins+k]*phase;
		
	}
	return thrust::norm(accum);			
}

template <typename value_t, bool weighted>
__global__ void reconstruction_red(thrust::complex<value_t>* samples,
                                value_t* frequencies, value_t* time_delays,
                                value_t* phis, value_t* rec, value_t R,
                                value_t wmix, int bins, int grid_size, int N,
                                int packet)
{
	
	/*
	 * Fastest so far
	 * 
	 * */
	
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    
    //printf("id: %d\n", tid);

    if(tid<grid_size*grid_size*bins) {
		
		int k = tid%bins; //frequency
        int i = (tid/bins)%grid_size; //y
        int j = tid/(bins*grid_size); //x

        value_t x = (-1+(value_t)(2*j+1)/grid_size)*R; //coords[j];
        value_t y = (-1+(value_t)(2*i+1)/grid_size)*R; //coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {
			
			value_t fc = 2*M_PI*frequencies[k] + wmix; //maybe to shared memory
			//value_t f = wmix/2;
			value_t result;
            
            
            if(weighted) {
				result = weighted_beamforming(fc, N, grid_size, packet, bins,
												i, j, k, time_delays, phis, samples);
			} else {
				result = beamforming(fc, N, grid_size, packet, bins,
												i, j, k, time_delays, phis, samples);
			}
			rec[((packet*grid_size+i)*grid_size+j)*bins+k] = result;
        }

    }
}

template <typename value_t>
__global__ void reconstruction_red2(thrust::complex<value_t>* samples,
                                value_t* frequencies,
                                value_t* coords, value_t* rec, value_t R,
                                value_t wmix, int bins, int grid_size, int N)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<grid_size*grid_size*bins) {

        int i = tid%grid_size;
        int j = (tid/grid_size)%grid_size;
        int k = tid/(grid_size*grid_size);

        value_t x = coords[j];
        value_t y = coords[i];
        value_t d = x*x+y*y;

        if(d<=R*R) {
			
			value_t f = frequencies[k]; //maybe to shared memory

            thrust::complex<value_t> accum(0);

            for(int l=0; l<N; ++l) {
				value_t angle = frequencies[l]; //(value_t)l/N*2*M_PI;
				value_t xi = R*cos(angle);
				value_t yi = R*sin(angle);
				
				value_t dx = xi-x;
				value_t dy = yi-y;
				value_t dist = sqrt(dx*dx + dy*dy);
				
				value_t time_delay = dist/SPEED_OF_LIGHT;
				value_t phi_c = atan2(-dy, -dx) + M_PI;
				
				//value_t phi = time_delays[(l*grid_size+j)*grid_size+i]*(2*M_PI*f+wmix);
				//phi += phis[(l*grid_size+j)*grid_size+i];
				
				value_t phi = time_delay*(2*M_PI*f+wmix)+phi_c;
				thrust::complex<value_t> phase(cos(phi), sin(phi));
                accum += samples[l*bins+k]*phase;
            }
            rec[(grid_size*i+j)*bins+k] = thrust::abs(accum);

        }

    }
}

template <typename value_t>
Reconstruction_GPU<value_t>::Reconstruction_GPU(int grid_size, int n_packets,
						arma::Col<value_t> frequency,
						const Antenna_Array<value_t>& array, bool weighted):
Reconstruction<value_t>(grid_size, n_packets, frequency, array, weighted)
{
	init_gpu();
}

template <typename value_t>
Reconstruction_GPU<value_t>::~Reconstruction_GPU()
{
	FREE_CUDA(frequencies_D);
	FREE_CUDA(time_delays_D);
	FREE_CUDA(phis_D);
	FREE_CUDA(samples_D);
	FREE_CUDA(reconstructed_D);
	
	FREE_HOST(samples_H);
	//FREE_HOST(reconstructed_H);
	free(reconstructed_H);
}

/*
template <typename value_t>
void Reconstruction_GPU<value_t>::calc_phase(
                        const std::vector<arma::Mat<value_t>>& grid_time_delays,
                        const std::vector<arma::Mat<value_t>>& grid_phis,
                        std::complex<value_t>* const grid_phase)
{
	
	int bins = frequency.n_elem;

    for(int i=0; i<N; ++i) {
		arma::Mat<value_t> time_delay = grid_time_delays[i].t();
		arma::Mat<value_t> phi_c = grid_phis[i].t();
        for(int l=0; l<bins; ++l) {
			value_t f = frequency(l);
            for(int k=0; k<grid_size; ++k) {
                for(int j=0; j<grid_size; ++j) {
                    //~ value_t phi = grid_time_delays[i](j,k)*(2*M_PI*frequency(l)+wmix);
                    //~ phi += grid_phis[i](j,k);
                    value_t phi = time_delay(j,k)*(2*M_PI*f+wmix);
                    phi += phi_c(j,k);
                    //grid_phase[((grid_size*j+k)*frequency.n_elem+l)*N+i] = std::complex<value_t>(cos(phi), sin(phi));
					grid_phase[((i*bins+l)*grid_size+k)*grid_size+j] = std::complex<value_t>(cos(phi), sin(phi));
                }
            }
        }
    }
} */

template <typename value_t>
unsigned int Reconstruction_GPU<value_t>::get_max_bin(unsigned int packet)
{
	
	int bins = this->bins;
	int grid_size = this->grid_size;

	value_t max_val = std::numeric_limits<value_t>::min();

	unsigned int index;
	//int packet = 0;
	for(unsigned int i=0; i<bins; ++i) {
		for(unsigned int j=0; j<grid_size; ++j) {
			for(unsigned int l=0; l<grid_size; ++l) {
				
				value_t val = reconstructed_H[((packet*grid_size+j)*grid_size+l)*bins+i];
				
				//std::cerr << i << " " << val << " " << max_val << std::endl;
				if(val>max_val && !std::isinf(val)) {
					max_val=val;
					index = i;
				}
			}
		}
	}

	//std::cerr << "Max frequency: " << this->frequency[index] 
	//			<< " (bin: " << index << ") val: " << max_val << " packet: "
	//			<< packet << std::endl;
				
	/*if(index>bins) {
		for(unsigned int i=0; i<bins; ++i) {
			for(unsigned int j=0; j<grid_size; ++j) {
				for(unsigned int l=0; l<grid_size; ++l) {
					
					value_t val = reconstructed_H[((packet*grid_size+j)*grid_size+l)*bins+i];
					
					std::cerr << i << " " << j << " " << l << " "
								<< val << " " << max_val << std::endl;
				}
			}
		}
	}*/

	return index;
}

template <typename value_t>
void Reconstruction_GPU<value_t>::init_gpu()
{
	
	std::cerr << "init" << std::endl;
	
	int N = this->N;
	int bins = this->bins;
	int n_packets = this->n_packets;
	int grid_size = this->grid_size;

    thrust::host_vector<value_t> time_delays_H(N*grid_size*grid_size);	CUERR
    thrust::host_vector<value_t> phis_H(N*grid_size*grid_size);			CUERR

	for(int l=0; l<N; ++l) {
		/*arma::Mat<value_t> mat_delay= this->grid_time_delays[i].t();
		arma::Mat<value_t> mat_phi = this->grid_phis[i].t();
		thrust::copy(mat_delay.begin(),mat_delay.end(),
					time_delays_H.begin()+i*grid_size*grid_size);		CUERR
		thrust::copy(mat_phi.begin(),mat_phi.end(),
					phis_H.begin()+i*grid_size*grid_size);				CUERR*/
		for(int i=0; i<grid_size; ++i) {
			for(int j=0; j<grid_size; ++j) {
				value_t time_delay = this->grid_time_delays[l](j,i);
				value_t phi = this->grid_phis[l](j,i);
				time_delays_H[(j*grid_size+i)*N+l] = time_delay;
				phis_H[(j*grid_size+i)*N+l] = phi;
			}
		}
	}
	
	hipHostMalloc(&(this->samples_H), 
				n_packets*N*bins*sizeof(thrust::complex<value_t>));		CUERR
	std::cerr << "allocating " 
				<< (grid_size*grid_size*bins*n_packets*sizeof(value_t)/1e9) 
				<< "GB of unpinned host memory" << std::endl;
	//using pinned memory results in frequent errors here for whatever reason ...
	this->reconstructed_H=(value_t*) malloc(grid_size*grid_size*bins*n_packets*sizeof(value_t));
	//hipHostMalloc(&(this->reconstructed_H), 
	//			grid_size*grid_size*bins*n_packets*sizeof(value_t));	CUERR
				
	hipMalloc(&(this->samples_D), 
				n_packets*N*bins*sizeof(thrust::complex<value_t>));		CUERR
	hipMalloc(&(this->reconstructed_D), 
				grid_size*grid_size*bins*n_packets*sizeof(value_t));	CUERR
				
	hipMalloc(&(this->frequencies_D), bins*sizeof(value_t)); 			CUERR
	hipMalloc(&(this->time_delays_D), 
					N*grid_size*grid_size*sizeof(value_t)); 			CUERR
	hipMalloc(&(this->phis_D), N*grid_size*grid_size*sizeof(value_t)); CUERR


	hipMemcpy(this->time_delays_D, time_delays_H.data(), 
						time_delays_H.size()*sizeof(value_t), H2D);		CUERR									
	hipMemcpy(this->phis_D, phis_H.data(), 
						phis_H.size()*sizeof(value_t), H2D);			CUERR						
	hipMemcpy(this->frequencies_D, this->frequency.memptr(), 
						this->frequency.size()*sizeof(value_t), H2D);	CUERR
}

//~ template <typename value_t>
//~ void Reconstruction_GPU<value_t>::free_memory()
//~ {
	//~ //FREE_CUDA(grid_phase_D);
	
	//~ FREE_CUDA(frequencies_D);
	//~ FREE_CUDA(time_delays_D);
	//~ FREE_CUDA(phis_D);
	//~ FREE_CUDA(samples_D);
	//~ FREE_CUDA(reconstructed_D);
	
	//~ FREE_HOST(samples_H);
	//~ FREE_HOST(reconstructed_H);
//~ }

template <typename value_t>
arma::Mat<value_t> Reconstruction_GPU<value_t>::get_img(unsigned int packet, 
															unsigned int bin)
{
	//int packet=0;
	
	if(bin >= this->frequency.n_elem) {
		std::string err = "Bin " + std::to_string(bin) 
								+ " is not a valid frequency bin!";
		throw std::out_of_range(err);
	}
	
	//std::cerr << "Fetching image for packet " << packet << " and bin " << bin << std::endl;

	int grid_size = this->grid_size;
	int bins = this->bins;
	arma::Mat<value_t> img(grid_size, grid_size);

	for(int i=0; i<grid_size; ++i) {
		for(int j=0; j<grid_size; ++j) {
			img(j,i) = reconstructed_H[((packet*grid_size+i)*grid_size+j)*bins+bin];

		}
	}

	return img;
}

template <typename value_t>
void Reconstruction_GPU<value_t>::print(unsigned int packet)
{
	int grid_size = this->grid_size;
	int bins = this->bins;
	arma::Mat<value_t> img(grid_size, grid_size);
	
	for(int k=0; k<bins; ++k) {
		for(int i=0; i<grid_size; ++i) {
			for(int j=0; j<grid_size; ++j) {
				std::cout << reconstructed_H[((packet*grid_size+i)*grid_size+j)*bins+k] << ' ';
			}
			std::cout << '\n';
		}
	}

}

template <typename value_t>
void Reconstruction_GPU<value_t>::run(const std::vector<std::complex<value_t>>& samples)
{
	std::cerr << "run" << std::endl;
		
	int N = this->N;
	int bins = this->bins;
	int n_packets = this->n_packets;
	int grid_size = this->grid_size;
	
	//~ int bins_ = samples[0][0].frequency.n_elem;
    //~ int n_packets_ = samples[0].size();
	//~ int N_ = samples.size();
	
	//~ if(N!=N_ || bins != bins_ || n_packets != n_packets_)
        //~ throw std::invalid_argument(
				//~ "'samples' input array dimension is (" + std::to_string(N_) 
				//~ + "," + std::to_string(n_packets_) + "," 
				//~ + std::to_string(bins_) + ") but expected dimension was ("
				//~ + std::to_string(N) + "," + std::to_string(n_packets) + ","
				//~ + std::to_string(bins) + ")" );

	size_t rec_size = grid_size*grid_size*bins*n_packets;
	size_t samples_size = n_packets*N*bins;
	
    TIMERSTART(REC)
    								
	std::cerr << "Copy CPU" << std::endl;
	size_t memsize_samples=sizeof(thrust::complex<value_t>)*samples_size;
	

    TIMERSTART(COPY_CPU)
    //~ for(int j=0; j<n_packets; ++j) {
		//~ for(int i=0; i<N; ++i) {
			//~ //thrust::copy(samples[i][j].frequency_data.begin(),
			//~ //				samples[i][j].frequency_data.end(),
			//~ //				samples_H+(j*N+i)*bins);					CUERR
			//~ hipMemcpy(this->samples_H, samples[i][j].frequency_data.memptr(), 
							//~ bins*sizeof(thrust::complex<value_t>), H2H);	CUERR
		//~ }
	//~ }
	hipMemcpy(this->samples_H, samples.data(), 
											memsize_samples, H2H);			CUERR	
    //TIMERSTOP(COPY_CPU)
    TIMERBW(memsize_samples, COPY_CPU)

    //copy data to GPU
    
    std::cerr << "Copy GPU" << std::endl;
    TIMERSTART(COPY_GPU)
    
	hipMemcpy(this->samples_D, this->samples_H, 
											memsize_samples, H2D);			CUERR	


    //TIMERSTOP(COPY_GPU)
    TIMERBW(memsize_samples, COPY_GPU)
	
	thrust::device_ptr<value_t> rec_thrust =
									thrust::device_pointer_cast(reconstructed_D);  
    thrust::fill(rec_thrust, rec_thrust+rec_size, value_t(-1));			CUERR
	
    int threads = 512;
    int tasks=grid_size*grid_size*bins;
    int blocks = SDIV(tasks, threads);

    //~ TIMERSTART(KERNEL)
    //~ reconstruction<<<blocks, threads>>>(samples_dev,
                                        //~ (thrust::complex<value_t>*)grid_phase,
                                        //~ coords_dev, rec_dev, grid.R,
                                        //~ bins, grid_size, N);   CUERR
    //~ TIMERSTOP(KERNEL)
    
    std::cerr << "start kernel" << std::endl;
    TIMERSTART(KERNELS)
    if(this->weighted) {
		for(int packet=0; packet<n_packets; ++packet) {
			reconstruction_red<value_t, true><<<blocks, threads>>>((thrust::complex<value_t>*)samples_D,
												frequencies_D, time_delays_D,
												phis_D, reconstructed_D, this->R,
												this->wmix, bins, grid_size, N, packet);   CUERR
		}
	} else {
		for(int packet=0; packet<n_packets; ++packet) {
			reconstruction_red<value_t, false><<<blocks, threads>>>((thrust::complex<value_t>*)samples_D,
												frequencies_D, time_delays_D,
												phis_D, reconstructed_D, this->R,
												this->wmix, bins, grid_size, N, packet);   CUERR
		}
	}
	TIMERSTOP(KERNELS)

    //copy back result
    size_t memsize_res = rec_size*sizeof(value_t);
    TIMERSTART(COPY_BACK)
    //thrust::copy(reconstructed_D.begin(), reconstructed_D.end(),
    //                reconstructed.begin());                             CUERR
	hipMemcpy(this->reconstructed_H, this->reconstructed_D, 
											memsize_res, D2H);			CUERR	
	//TIMERSTOP(COPY_BACK)
	TIMERBW(memsize_res, COPY_BACK)

    TIMERSTOP(REC)
}

DEFINE_TEMPLATES(Reconstruction_GPU)

